#include "hip/hip_runtime.h"
#include "../include/nested_dissection.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <algorithm>
#include <random>
#include <cassert>
#include <numeric>
#include <chrono>
#include <map>
#include <set>
#include <cmath>
#include <climits>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// Enhanced GPU kernels for heavy edge matching and graph coarsening
// Add these to your nested_dissection.cu file

// Additional kernel for counting coarse vertex degrees
__global__ void count_coarse_vertex_degrees(int *row_ptr, int *col_idx, int *vertex_map,
                                            int *degree_count, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        int coarse_v = vertex_map[idx];

        for (int e = row_ptr[idx]; e < row_ptr[idx + 1]; e++)
        {
            int neighbor = col_idx[e];
            int coarse_neighbor = vertex_map[neighbor];

            if (coarse_v != coarse_neighbor)
            {
                atomicAdd(&degree_count[coarse_v], 1);
            }
        }
    }
}

__global__ void initialize_matching_arrays(int *match, int *vertex_weights, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        match[idx] = -1;
        vertex_weights[idx] = 1;
    }
}

__global__ void heavy_edge_matching_phase1(int *row_ptr, int *col_idx, int *edge_weights,
                                           int *match, int *proposals, hiprandState *states, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n && match[idx] == -1)
    {
        int best_neighbor = -1;
        int max_weight = -1;

        // Find heaviest unmatched neighbor
        for (int e = row_ptr[idx]; e < row_ptr[idx + 1]; e++)
        {
            int neighbor = col_idx[e];
            int weight = edge_weights[e];

            if (neighbor != idx && match[neighbor] == -1 && weight > max_weight)
            {
                max_weight = weight;
                best_neighbor = neighbor;
            }
        }

        proposals[idx] = best_neighbor;
    }
    else
    {
        proposals[idx] = -1;
    }
}

__global__ void heavy_edge_matching_phase2(int *proposals, int *match, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        int proposed_neighbor = proposals[idx];
        if (proposed_neighbor != -1 && proposals[proposed_neighbor] == idx)
        {
            // Mutual proposal - create match
            if (idx < proposed_neighbor)
            {
                match[idx] = proposed_neighbor;
                match[proposed_neighbor] = idx;
            }
        }
    }
}

__global__ void compute_coarse_vertex_mapping(int *match, int *vertex_map, int *coarse_vertex_count, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        if (match[idx] == -1)
        {
            // Unmatched vertex gets its own new vertex ID
            vertex_map[idx] = atomicAdd(coarse_vertex_count, 1);
        }
        else if (idx < match[idx])
        {
            // For matched pair, smaller ID gets the new vertex ID
            int new_id = atomicAdd(coarse_vertex_count, 1);
            vertex_map[idx] = new_id;
            vertex_map[match[idx]] = new_id;
        }
    }
}

__global__ void count_coarse_edges(int *row_ptr, int *col_idx, int *vertex_map,
                                   int *coarse_edge_count, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        int coarse_v = vertex_map[idx];

        for (int e = row_ptr[idx]; e < row_ptr[idx + 1]; e++)
        {
            int neighbor = col_idx[e];
            int coarse_neighbor = vertex_map[neighbor];

            if (coarse_v != coarse_neighbor)
            {
                atomicAdd(coarse_edge_count, 1);
            }
        }
    }
}

__global__ void build_coarse_graph_structure(int *old_row_ptr, int *old_col_idx, int *old_edge_weights,
                                             int *vertex_map, int old_n, int coarse_n,
                                             int *new_row_ptr, int *edge_positions, int *temp_edges,
                                             int *temp_weights, int *temp_neighbors)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < old_n)
    {
        int coarse_v = vertex_map[idx];

        for (int e = old_row_ptr[idx]; e < old_row_ptr[idx + 1]; e++)
        {
            int neighbor = old_col_idx[e];
            int coarse_neighbor = vertex_map[neighbor];
            int weight = old_edge_weights[e];

            if (coarse_v != coarse_neighbor)
            {
                int pos = atomicAdd(&edge_positions[coarse_v], 1);
                int global_pos = new_row_ptr[coarse_v] + pos;

                temp_neighbors[global_pos] = coarse_neighbor;
                temp_weights[global_pos] = weight;
            }
        }
    }
}

__global__ void merge_parallel_edges(int *row_ptr, int *col_idx, int *edge_weights,
                                     int *new_col_idx, int *new_edge_weights, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        int start = row_ptr[idx];
        int end = row_ptr[idx + 1];
        int degree = end - start;

        if (degree > 0)
        {
            // Sort edges by neighbor ID (simple bubble sort for small degrees)
            for (int i = 0; i < degree - 1; i++)
            {
                for (int j = start; j < start + degree - 1 - i; j++)
                {
                    if (col_idx[j] > col_idx[j + 1])
                    {
                        // Swap neighbors
                        int temp_neighbor = col_idx[j];
                        col_idx[j] = col_idx[j + 1];
                        col_idx[j + 1] = temp_neighbor;

                        // Swap weights
                        int temp_weight = edge_weights[j];
                        edge_weights[j] = edge_weights[j + 1];
                        edge_weights[j + 1] = temp_weight;
                    }
                }
            }

            // Merge parallel edges
            int write_pos = start;
            for (int read_pos = start; read_pos < end; read_pos++)
            {
                if (write_pos == start || col_idx[read_pos] != col_idx[write_pos - 1])
                {
                    // New unique neighbor
                    col_idx[write_pos] = col_idx[read_pos];
                    edge_weights[write_pos] = edge_weights[read_pos];
                    write_pos++;
                }
                else
                {
                    // Parallel edge - add weight to previous
                    edge_weights[write_pos - 1] += edge_weights[read_pos];
                }
            }

            // Update row pointer for next vertex
            if (idx < n - 1)
            {
                row_ptr[idx + 1] = write_pos;
            }
        }
    }
}

__global__ void compute_vertex_weights_coarse(int *match, int *old_vertex_weights,
                                              int *new_vertex_weights, int *vertex_map, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        int coarse_v = vertex_map[idx];
        atomicAdd(&new_vertex_weights[coarse_v], old_vertex_weights[idx]);
    }
}

// Utility kernel for parallel prefix sum (simplified version)
__global__ void parallel_prefix_sum(int *input, int *output, int n)
{
    extern __shared__ int temp[];
    int tid = threadIdx.x;
    int offset = 1;

    // Load input into shared memory
    if (blockIdx.x * blockDim.x + tid < n)
    {
        temp[2 * tid] = input[2 * blockIdx.x * blockDim.x + 2 * tid];
        temp[2 * tid + 1] = input[2 * blockIdx.x * blockDim.x + 2 * tid + 1];
    }
    else
    {
        temp[2 * tid] = 0;
        temp[2 * tid + 1] = 0;
    }

    // Build sum in place up the tree
    for (int d = blockDim.x; d > 0; d >>= 1)
    {
        __syncthreads();
        if (tid < d)
        {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    // Clear the last element
    if (tid == 0)
        temp[2 * blockDim.x - 1] = 0;

    // Traverse down tree & build scan
    for (int d = 1; d < 2 * blockDim.x; d *= 2)
    {
        offset >>= 1;
        __syncthreads();
        if (tid < d)
        {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    // Write results to device memory
    if (2 * blockIdx.x * blockDim.x + 2 * tid < n)
    {
        output[2 * blockIdx.x * blockDim.x + 2 * tid] = temp[2 * tid];
    }
    if (2 * blockIdx.x * blockDim.x + 2 * tid + 1 < n)
    {
        output[2 * blockIdx.x * blockDim.x + 2 * tid + 1] = temp[2 * tid + 1];
    }
}

template <typename RandomIt>
void simple_random_shuffle(RandomIt first, RandomIt last)
{
    std::random_device rd;
    std::mt19937 g(rd());
    std::shuffle(first, last, g);
}

// CUDA kernel implementations
__global__ void init_random_states(hiprandState *states, int n, unsigned long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

__global__ void compute_vertex_degrees(int *row_ptr, int n, int *degrees)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        degrees[idx] = row_ptr[idx + 1] - row_ptr[idx];
    }
}

__global__ void heavy_edge_matching_kernel(int *row_ptr, int *col_idx, int *edge_weights,
                                           int *match, hiprandState *states, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n && match[idx] == -1)
    {
        int best_neighbor = -1;
        int max_weight = -1;

        // Find heaviest edge
        for (int e = row_ptr[idx]; e < row_ptr[idx + 1]; e++)
        {
            int neighbor = col_idx[e];
            if (neighbor != idx && match[neighbor] == -1 && edge_weights[e] > max_weight)
            {
                max_weight = edge_weights[e];
                best_neighbor = neighbor;
            }
        }

        // Try to match with best neighbor
        if (best_neighbor != -1)
        {
            int old = atomicCAS(&match[best_neighbor], -1, idx);
            if (old == -1)
            {
                match[idx] = best_neighbor;
            }
        }
    }
}

__global__ void contract_graph_kernel(int *old_row_ptr, int *old_col_idx, int *old_edge_weights,
                                      int *match, int *vertex_map, int old_n,
                                      int *new_row_ptr, int *new_col_idx, int *new_edge_weights,
                                      int *new_vertex_weights, int new_n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < old_n)
    {
        // Simplified contraction - in practice this would be more complex
        int new_vertex = vertex_map[idx];
        if (new_vertex < new_n)
        {
            atomicAdd(&new_vertex_weights[new_vertex], 1);
        }
    }
}

// Constructor
ImprovedGPUNestedDissection::ImprovedGPUNestedDissection(int max_v) : max_vertices(max_v)
{
    // Allocate device memory
    hipMalloc(&d_partition, max_vertices * sizeof(int));
    hipMalloc(&d_degrees, max_vertices * sizeof(int));
    hipMalloc(&d_match, max_vertices * sizeof(int));
    hipMalloc(&d_vertex_map, max_vertices * sizeof(int));
    hipMalloc(&d_temp_array, max_vertices * sizeof(int));
    hipMalloc(&d_rand_states, max_vertices * sizeof(hiprandState));

    // Initialize random states
    dim3 grid((max_vertices + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 block(BLOCK_SIZE);
    init_random_states<<<grid, block>>>(d_rand_states, max_vertices,
                                        std::chrono::high_resolution_clock::now().time_since_epoch().count());
    hipDeviceSynchronize();
}

// Destructor
ImprovedGPUNestedDissection::~ImprovedGPUNestedDissection()
{
    hipFree(d_partition);
    hipFree(d_degrees);
    hipFree(d_match);
    hipFree(d_vertex_map);
    hipFree(d_temp_array);
    hipFree(d_rand_states);
}

// Main compute ordering method
// Improved Compute Ordering Function
// Replace the existing compute_ordering function in nested_dissection.cu

std::vector<int> ImprovedGPUNestedDissection::compute_ordering(const Graph &graph)
{
    h_ordering.clear();
    h_ordering.reserve(graph.n_vertices);

    std::vector<int> separator_stack;
    std::vector<int> all_vertices(graph.n_vertices);
    std::iota(all_vertices.begin(), all_vertices.end(), 0);

    std::queue<DissectionTask> task_queue;
    task_queue.emplace(all_vertices, 0);

    int max_levels = static_cast<int>(std::log2(graph.n_vertices)) + 2;

    while (!task_queue.empty())
    {
        DissectionTask task = task_queue.front();
        task_queue.pop();

        std::cout << "Processing task with " << task.vertices.size()
                  << " vertices at level " << task.level << std::endl;

        if (task.level > max_levels || task.vertices.size() <= MIN_SUBGRAPH_SIZE)
        {
            // Base case: add vertices to ordering
            for (int v : task.vertices)
            {
                h_ordering.push_back(v);
            }
            continue;
        }

        // Use multilevel approach for larger subgraphs (lowered threshold)
        if (task.vertices.size() > 200)
        { // Reduced from 1000 to 200
            std::cout << "  Attempting multilevel partition..." << std::endl;
            Partition partition = multilevel_partition(graph, task.vertices);
            partition.print_stats();

            // Verify partition quality
            int total_vertices = partition.left_vertices.size() +
                                 partition.right_vertices.size() +
                                 partition.separator_vertices.size();

            if (total_vertices == task.vertices.size())
            {
                // Partition is valid, use it
                std::cout << "  Multilevel partition successful" << std::endl;

                // Store separators for later
                for (int v : partition.separator_vertices)
                {
                    separator_stack.push_back(v);
                }

                // Add subtasks
                if (!partition.left_vertices.empty())
                {
                    task_queue.emplace(partition.left_vertices, task.level + 1);
                }
                if (!partition.right_vertices.empty())
                {
                    task_queue.emplace(partition.right_vertices, task.level + 1);
                }
            }
            else
            {
                // Partition failed, fall back to direct
                std::cout << "  Multilevel partition invalid, using direct" << std::endl;
                Partition direct_partition_result = direct_partition(graph, task.vertices);
                direct_partition_result.print_stats();

                for (int v : direct_partition_result.separator_vertices)
                {
                    separator_stack.push_back(v);
                }

                if (!direct_partition_result.left_vertices.empty())
                {
                    task_queue.emplace(direct_partition_result.left_vertices, task.level + 1);
                }
                if (!direct_partition_result.right_vertices.empty())
                {
                    task_queue.emplace(direct_partition_result.right_vertices, task.level + 1);
                }
            }
        }
        else
        {
            // Use direct partitioning for smaller graphs
            std::cout << "  Using direct partitioning" << std::endl;
            Partition partition = direct_partition(graph, task.vertices);
            partition.print_stats();

            for (int v : partition.separator_vertices)
            {
                separator_stack.push_back(v);
            }

            if (!partition.left_vertices.empty())
            {
                task_queue.emplace(partition.left_vertices, task.level + 1);
            }
            if (!partition.right_vertices.empty())
            {
                task_queue.emplace(partition.right_vertices, task.level + 1);
            }
        }
    }

    // Add separators at the end in reverse order (typical nested dissection)
    std::reverse(separator_stack.begin(), separator_stack.end());
    for (int v : separator_stack)
    {
        h_ordering.push_back(v);
    }

    std::cout << "Final ordering: " << h_ordering.size() << " vertices" << std::endl;

    // Verify ordering completeness
    if (h_ordering.size() != graph.n_vertices)
    {
        std::cerr << "Warning: Incomplete ordering! Expected " << graph.n_vertices
                  << ", got " << h_ordering.size() << std::endl;
    }

    return h_ordering;
}

// Improved Multilevel Partitioning
// Replace the existing multilevel_partition function in nested_dissection.cu

// Improved Multilevel Partitioning
// Replace the existing multilevel_partition function in nested_dissection.cu

Partition ImprovedGPUNestedDissection::multilevel_partition(const Graph &original, const std::vector<int> &vertices)
{
    std::cout << "  Using multilevel partitioning" << std::endl;

    // Safety check: if subgraph is too small, use direct partitioning
    if (vertices.size() < 100)
    {
        std::cout << "  (Too small for multilevel, using direct)" << std::endl;
        return direct_partition(original, vertices);
    }

    try
    {
        // Extract subgraph
        Graph subgraph = extract_subgraph(original, vertices);

        // Coarsening phase - use heavy edge matching
        std::vector<Graph> graph_hierarchy;
        std::vector<std::vector<int>> vertex_mappings;

        // Store the initial subgraph
        graph_hierarchy.push_back(std::move(subgraph));

        // Coarsen until small enough or no more reduction possible
        int coarsening_levels = 0;
        const int max_coarsening_levels = 10;

        while (graph_hierarchy.back().n_vertices > 50 && coarsening_levels < max_coarsening_levels)
        {
            auto [coarse_graph, mapping] = coarsen_graph(graph_hierarchy.back());

            // Check if we got sufficient reduction
            double reduction_ratio = (double)coarse_graph.n_vertices / graph_hierarchy.back().n_vertices;
            if (reduction_ratio > 0.85)
            {
                // Not enough reduction, stop coarsening
                std::cout << "  Insufficient reduction (" << (reduction_ratio * 100)
                          << "%), stopping coarsening" << std::endl;
                break;
            }

            vertex_mappings.push_back(mapping);
            graph_hierarchy.push_back(std::move(coarse_graph));
            coarsening_levels++;

            std::cout << "  Coarsening level " << coarsening_levels
                      << ": " << graph_hierarchy.back().n_vertices << " vertices" << std::endl;
        }

        // Initial partition of coarsest graph
        Partition coarse_partition = initial_partition(graph_hierarchy.back());
        std::cout << "  Initial partition on coarsest graph completed" << std::endl;

        // Uncoarsening and refinement phase
        for (int i = graph_hierarchy.size() - 2; i >= 0; i--)
        {
            std::cout << "  Projecting to level " << i
                      << " (" << graph_hierarchy[i].n_vertices << " vertices)" << std::endl;
            std::cout << "  Current partition sizes: L=" << coarse_partition.left_vertices.size()
                      << " R=" << coarse_partition.right_vertices.size()
                      << " S=" << coarse_partition.separator_vertices.size() << std::endl;

            // Safety check before projection
            if (i < vertex_mappings.size())
            {
                std::cout << "  Vertex mapping size: " << vertex_mappings[i].size() << std::endl;
                coarse_partition = project_and_refine(graph_hierarchy[i], coarse_partition,
                                                      vertex_mappings[i]);
            }
            else
            {
                std::cout << "  ERROR: Invalid mapping index " << i << std::endl;
                break;
            }
        }

        // Map back to original vertex IDs
        Partition final_partition;
        std::cout << "  Mapping back to original vertices (size: " << vertices.size() << ")" << std::endl;

        for (int v : coarse_partition.left_vertices)
        {
            if (v >= 0 && v < vertices.size())
            {
                final_partition.left_vertices.push_back(vertices[v]);
            }
            else
            {
                std::cout << "  WARNING: Invalid left vertex index " << v << std::endl;
            }
        }
        for (int v : coarse_partition.right_vertices)
        {
            if (v >= 0 && v < vertices.size())
            {
                final_partition.right_vertices.push_back(vertices[v]);
            }
            else
            {
                std::cout << "  WARNING: Invalid right vertex index " << v << std::endl;
            }
        }
        for (int v : coarse_partition.separator_vertices)
        {
            if (v >= 0 && v < vertices.size())
            {
                final_partition.separator_vertices.push_back(vertices[v]);
            }
            else
            {
                std::cout << "  WARNING: Invalid separator vertex index " << v << std::endl;
            }
        }

        final_partition.edge_cut = coarse_partition.edge_cut;
        final_partition.balance_ratio = coarse_partition.balance_ratio;

        std::cout << "  Multilevel partitioning completed successfully" << std::endl;
        return final_partition;
    }
    catch (const std::exception &e)
    {
        std::cout << "  Multilevel partitioning failed: " << e.what() << std::endl;
        std::cout << "  Falling back to direct partitioning" << std::endl;
        return direct_partition(original, vertices);
    }
}

// Fallback direct partitioning (improved geometric + graph-based)
Partition ImprovedGPUNestedDissection::direct_partition(const Graph &graph, const std::vector<int> &vertices)
{
    std::cout << "  Using direct partitioning" << std::endl;

    Graph subgraph = extract_subgraph(graph, vertices);
    return graph_based_partition(subgraph, vertices);
}

// Heavy edge matching for coarsening (currently simplified to avoid segfaults)

// Replace the existing coarsen_graph function in nested_dissection.cu
std::pair<Graph, std::vector<int>> ImprovedGPUNestedDissection::coarsen_graph(const Graph &graph)
{
    const int n = graph.n_vertices;
    const dim3 block(BLOCK_SIZE);
    const dim3 grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Allocate device memory for matching
    int *d_proposals, *d_coarse_vertex_count, *d_coarse_edge_count;
    hipMalloc(&d_proposals, n * sizeof(int));
    hipMalloc(&d_coarse_vertex_count, sizeof(int));
    hipMalloc(&d_coarse_edge_count, sizeof(int));

    // Initialize matching arrays
    initialize_matching_arrays<<<grid, block>>>(d_match, d_temp_array, n);
    hipDeviceSynchronize();

    // Phase 1: Each vertex proposes to its heaviest neighbor
    heavy_edge_matching_phase1<<<grid, block>>>(
        graph.row_ptr, graph.col_idx, graph.edge_weights,
        d_match, d_proposals, d_rand_states, n);
    hipDeviceSynchronize();

    // Phase 2: Create matches from mutual proposals
    heavy_edge_matching_phase2<<<grid, block>>>(d_proposals, d_match, n);
    hipDeviceSynchronize();

    // Initialize counters
    hipMemset(d_coarse_vertex_count, 0, sizeof(int));
    hipMemset(d_coarse_edge_count, 0, sizeof(int));

    // Compute vertex mapping and count coarse vertices
    compute_coarse_vertex_mapping<<<grid, block>>>(d_match, d_vertex_map, d_coarse_vertex_count, n);
    hipDeviceSynchronize();

    // Get coarse vertex count
    int coarse_n;
    hipMemcpy(&coarse_n, d_coarse_vertex_count, sizeof(int), hipMemcpyDeviceToHost);

    if (coarse_n <= 0 || coarse_n >= n)
    {
        // Coarsening failed, return original graph
        std::cout << "  Coarsening failed, returning original graph" << std::endl;
        hipFree(d_proposals);
        hipFree(d_coarse_vertex_count);
        hipFree(d_coarse_edge_count);

        // Create identity mapping
        std::vector<int> identity_map(n);
        std::iota(identity_map.begin(), identity_map.end(), 0);

        // Deep copy the original graph
        Graph copy_graph(n, graph.n_edges);
        hipMemcpy(copy_graph.row_ptr, graph.row_ptr, (n + 1) * sizeof(int), hipMemcpyDeviceToDevice);
        hipMemcpy(copy_graph.col_idx, graph.col_idx, graph.n_edges * sizeof(int), hipMemcpyDeviceToDevice);
        hipMemcpy(copy_graph.edge_weights, graph.edge_weights, graph.n_edges * sizeof(int), hipMemcpyDeviceToDevice);
        hipMemcpy(copy_graph.vertex_weights, graph.vertex_weights, n * sizeof(int), hipMemcpyDeviceToDevice);

        return {std::move(copy_graph), identity_map};
    }

    // Count edges in coarse graph
    count_coarse_edges<<<grid, block>>>(graph.row_ptr, graph.col_idx, d_vertex_map, d_coarse_edge_count, n);
    hipDeviceSynchronize();

    int coarse_edges;
    hipMemcpy(&coarse_edges, d_coarse_edge_count, sizeof(int), hipMemcpyDeviceToHost);

    // Allocate memory for coarse graph
    int *d_new_row_ptr, *d_edge_positions, *d_temp_neighbors, *d_temp_weights;
    hipMalloc(&d_new_row_ptr, (coarse_n + 1) * sizeof(int));
    hipMalloc(&d_edge_positions, coarse_n * sizeof(int));
    hipMalloc(&d_temp_neighbors, coarse_edges * sizeof(int));
    hipMalloc(&d_temp_weights, coarse_edges * sizeof(int));

    // Initialize arrays
    hipMemset(d_new_row_ptr, 0, (coarse_n + 1) * sizeof(int));
    hipMemset(d_edge_positions, 0, coarse_n * sizeof(int));

    // First pass: count edges per coarse vertex
    int *d_degree_count;
    hipMalloc(&d_degree_count, coarse_n * sizeof(int));
    hipMemset(d_degree_count, 0, coarse_n * sizeof(int));

    // Count degrees for each coarse vertex
    count_coarse_vertex_degrees<<<grid, block>>>(
        graph.row_ptr, graph.col_idx, d_vertex_map, d_degree_count, n);
    hipDeviceSynchronize();

    // Compute prefix sum for row pointers using Thrust
    thrust::device_ptr<int> degree_ptr(d_degree_count);
    thrust::device_ptr<int> row_ptr(d_new_row_ptr + 1);
    thrust::inclusive_scan(degree_ptr, degree_ptr + coarse_n, row_ptr);

    // Build coarse graph structure
    build_coarse_graph_structure<<<grid, block>>>(
        graph.row_ptr, graph.col_idx, graph.edge_weights, d_vertex_map, n, coarse_n,
        d_new_row_ptr, d_edge_positions, d_temp_neighbors, d_temp_weights, d_temp_neighbors);
    hipDeviceSynchronize();

    // Create final coarse graph
    Graph coarse_graph(coarse_n, coarse_edges);

    // Copy row pointers
    hipMemcpy(coarse_graph.row_ptr, d_new_row_ptr, (coarse_n + 1) * sizeof(int), hipMemcpyDeviceToDevice);

    // Merge parallel edges and sort
    const dim3 coarse_grid((coarse_n + BLOCK_SIZE - 1) / BLOCK_SIZE);
    merge_parallel_edges<<<coarse_grid, block>>>(
        coarse_graph.row_ptr, d_temp_neighbors, d_temp_weights,
        coarse_graph.col_idx, coarse_graph.edge_weights, coarse_n);
    hipDeviceSynchronize();

    // Compute coarse vertex weights
    hipMemset(coarse_graph.vertex_weights, 0, coarse_n * sizeof(int));
    compute_vertex_weights_coarse<<<grid, block>>>(
        d_match, graph.vertex_weights, coarse_graph.vertex_weights, d_vertex_map, n);
    hipDeviceSynchronize();

    // Copy vertex mapping to host
    std::vector<int> vertex_mapping(n);
    hipMemcpy(vertex_mapping.data(), d_vertex_map, n * sizeof(int), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_proposals);
    hipFree(d_coarse_vertex_count);
    hipFree(d_coarse_edge_count);
    hipFree(d_new_row_ptr);
    hipFree(d_edge_positions);
    hipFree(d_temp_neighbors);
    hipFree(d_temp_weights);
    hipFree(d_degree_count);

    std::cout << "  GPU Coarsened from " << n << " to " << coarse_n << " vertices (reduction: "
              << (100.0 * (n - coarse_n) / n) << "%)" << std::endl;

    return {std::move(coarse_graph), vertex_mapping};
}

// Graph-based partitioning using BFS + edge cutting
Partition ImprovedGPUNestedDissection::graph_based_partition(const Graph &subgraph, const std::vector<int> &original_vertices)
{
    Partition result;
    int n = subgraph.n_vertices;

    if (n <= 2)
    {
        if (n == 1)
            result.left_vertices.push_back(original_vertices[0]);
        if (n == 2)
        {
            result.left_vertices.push_back(original_vertices[0]);
            result.right_vertices.push_back(original_vertices[1]);
        }
        return result;
    }

    // Extract subgraph adjacency from the actual subgraph structure
    std::vector<int> h_row_ptr(n + 1);
    std::vector<int> h_col_idx(subgraph.n_edges);
    std::vector<int> h_edge_weights(subgraph.n_edges);

    // Copy subgraph data to host for processing
    hipMemcpy(h_row_ptr.data(), subgraph.row_ptr,
               (n + 1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_col_idx.data(), subgraph.col_idx,
               subgraph.n_edges * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_edge_weights.data(), subgraph.edge_weights,
               subgraph.n_edges * sizeof(int), hipMemcpyDeviceToHost);

    // Build adjacency list from the actual subgraph
    std::vector<std::vector<int>> adj_list(n);
    for (int v = 0; v < n; v++)
    {
        for (int e = h_row_ptr[v]; e < h_row_ptr[v + 1]; e++)
        {
            int neighbor = h_col_idx[e];
            adj_list[v].push_back(neighbor);
        }
    }

    // Use BFS from multiple starting points to find good partition
    std::vector<int> best_partition(n, -1);
    int best_cut = INT_MAX;
    double best_balance = 1.0;

    // Try different starting vertices
    for (int start = 0; start < std::min(n, 4); start++)
    {
        std::vector<int> partition(n, -1);
        std::queue<int> bfs_queue;

        bfs_queue.push(start);
        partition[start] = 0;
        int left_count = 1;
        int target_size = n / 2;

        // BFS to grow one side
        while (!bfs_queue.empty() && left_count < target_size)
        {
            int v = bfs_queue.front();
            bfs_queue.pop();

            // Add unvisited neighbors to the same partition
            for (int neighbor : adj_list[v])
            {
                if (partition[neighbor] == -1 && left_count < target_size)
                {
                    partition[neighbor] = 0;
                    bfs_queue.push(neighbor);
                    left_count++;
                }
            }
        }

        // Assign remaining vertices to right partition
        for (int i = 0; i < n; i++)
        {
            if (partition[i] == -1)
            {
                partition[i] = 1;
            }
        }

        // Compute cut and balance
        int cut = 0;
        for (int i = 0; i < n; i++)
        {
            for (int neighbor : adj_list[i])
            {
                if (partition[i] != partition[neighbor])
                {
                    cut++;
                }
            }
        }
        cut /= 2; // Each edge counted twice

        int left_size = 0, right_size = 0;
        for (int i = 0; i < n; i++)
        {
            if (partition[i] == 0)
                left_size++;
            else
                right_size++;
        }

        double balance = abs(left_size - right_size) / (double)n;

        // Prefer better balance, then better cut
        if (balance < best_balance || (balance == best_balance && cut < best_cut))
        {
            best_cut = cut;
            best_balance = balance;
            best_partition = partition;
        }
    }

    // Find separator vertices (vertices with neighbors in both partitions)
    std::vector<bool> is_separator(n, false);
    for (int i = 0; i < n; i++)
    {
        bool has_left_neighbor = false, has_right_neighbor = false;

        for (int neighbor : adj_list[i])
        {
            if (best_partition[neighbor] == 0)
                has_left_neighbor = true;
            if (best_partition[neighbor] == 1)
                has_right_neighbor = true;
        }

        if (has_left_neighbor && has_right_neighbor)
        {
            is_separator[i] = true;
        }
    }

    // Build result - remove separator vertices from left/right partitions
    for (int i = 0; i < n; i++)
    {
        int orig_vertex = original_vertices[i];
        if (is_separator[i])
        {
            result.separator_vertices.push_back(orig_vertex);
        }
        else if (best_partition[i] == 0)
        {
            result.left_vertices.push_back(orig_vertex);
        }
        else
        {
            result.right_vertices.push_back(orig_vertex);
        }
    }

    result.edge_cut = best_cut;
    result.balance_ratio = best_balance;

    return result;
}

// Helper function to extract subgraph
Graph ImprovedGPUNestedDissection::extract_subgraph(const Graph &original, const std::vector<int> &vertices)
{
    // Create a mapping from old vertex IDs to new vertex IDs
    std::vector<int> vertex_map(original.n_vertices, -1);
    for (int i = 0; i < vertices.size(); i++)
    {
        vertex_map[vertices[i]] = i;
    }

    // Count edges in subgraph by examining original graph
    std::vector<int> h_row_ptr(original.n_vertices + 1);
    std::vector<int> h_col_idx(original.n_edges);
    std::vector<int> h_edge_weights(original.n_edges);

    // Copy original graph to host for processing
    hipMemcpy(h_row_ptr.data(), original.row_ptr,
               (original.n_vertices + 1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_col_idx.data(), original.col_idx,
               original.n_edges * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_edge_weights.data(), original.edge_weights,
               original.n_edges * sizeof(int), hipMemcpyDeviceToHost);

    // Count subgraph edges
    int sub_edges = 0;
    for (int v : vertices)
    {
        for (int e = h_row_ptr[v]; e < h_row_ptr[v + 1]; e++)
        {
            int neighbor = h_col_idx[e];
            if (vertex_map[neighbor] != -1)
            {
                sub_edges++;
            }
        }
    }

    // Create subgraph
    Graph subgraph(vertices.size(), sub_edges);
    std::vector<int> sub_row_ptr(vertices.size() + 1, 0);
    std::vector<int> sub_col_idx(sub_edges);
    std::vector<int> sub_edge_weights(sub_edges);
    std::vector<int> sub_vertex_weights(vertices.size(), 1);

    int edge_pos = 0;
    for (int i = 0; i < vertices.size(); i++)
    {
        int v = vertices[i];
        sub_row_ptr[i] = edge_pos;

        for (int e = h_row_ptr[v]; e < h_row_ptr[v + 1]; e++)
        {
            int neighbor = h_col_idx[e];
            if (vertex_map[neighbor] != -1)
            {
                sub_col_idx[edge_pos] = vertex_map[neighbor];
                sub_edge_weights[edge_pos] = h_edge_weights[e];
                edge_pos++;
            }
        }
    }
    sub_row_ptr[vertices.size()] = edge_pos;

    // Copy to device
    hipMemcpy(subgraph.row_ptr, sub_row_ptr.data(),
               (vertices.size() + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(subgraph.col_idx, sub_col_idx.data(),
               sub_edges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(subgraph.edge_weights, sub_edge_weights.data(),
               sub_edges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(subgraph.vertex_weights, sub_vertex_weights.data(),
               vertices.size() * sizeof(int), hipMemcpyHostToDevice);

    return subgraph;
}

// Initial partitioning for coarsest graph
Partition ImprovedGPUNestedDissection::initial_partition(const Graph &graph)
{
    // Improved initial partitioning for coarsest graph
    Partition result;

    // Copy graph data to host for initial partitioning
    std::vector<int> h_row_ptr(graph.n_vertices + 1);
    std::vector<int> h_col_idx(graph.n_edges);

    hipMemcpy(h_row_ptr.data(), graph.row_ptr,
               (graph.n_vertices + 1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_col_idx.data(), graph.col_idx,
               graph.n_edges * sizeof(int), hipMemcpyDeviceToHost);

    // Use BFS-based partitioning for better quality
    std::vector<int> partition(graph.n_vertices, -1);
    std::queue<int> bfs_queue;

    // Start from vertex with lowest degree (or random if all similar)
    int start_vertex = 0;
    int min_degree = h_row_ptr[1] - h_row_ptr[0];
    for (int i = 1; i < graph.n_vertices; i++)
    {
        int degree = h_row_ptr[i + 1] - h_row_ptr[i];
        if (degree < min_degree)
        {
            min_degree = degree;
            start_vertex = i;
        }
    }

    bfs_queue.push(start_vertex);
    partition[start_vertex] = 0;
    int left_count = 1;
    int target_size = graph.n_vertices / 2;

    // BFS to grow left partition
    while (!bfs_queue.empty() && left_count < target_size)
    {
        int v = bfs_queue.front();
        bfs_queue.pop();

        for (int e = h_row_ptr[v]; e < h_row_ptr[v + 1]; e++)
        {
            int neighbor = h_col_idx[e];
            if (partition[neighbor] == -1 && left_count < target_size)
            {
                partition[neighbor] = 0;
                bfs_queue.push(neighbor);
                left_count++;
            }
        }
    }

    // Assign remaining vertices to right partition
    for (int i = 0; i < graph.n_vertices; i++)
    {
        if (partition[i] == -1)
        {
            partition[i] = 1;
        }
    }

    // Find separator vertices
    std::vector<bool> is_separator(graph.n_vertices, false);
    for (int v = 0; v < graph.n_vertices; v++)
    {
        bool has_left = false, has_right = false;
        for (int e = h_row_ptr[v]; e < h_row_ptr[v + 1]; e++)
        {
            int neighbor = h_col_idx[e];
            if (partition[neighbor] == 0)
                has_left = true;
            if (partition[neighbor] == 1)
                has_right = true;
        }
        if (has_left && has_right)
        {
            is_separator[v] = true;
        }
    }

    // Build partition result
    for (int i = 0; i < graph.n_vertices; i++)
    {
        if (is_separator[i])
        {
            result.separator_vertices.push_back(i);
        }
        else if (partition[i] == 0)
        {
            result.left_vertices.push_back(i);
        }
        else
        {
            result.right_vertices.push_back(i);
        }
    }

    return result;
}

// Project partition to finer graph and refine
// Safer Project and Refine Function
// Replace the existing project_and_refine function in nested_dissection.cu

Partition ImprovedGPUNestedDissection::project_and_refine(const Graph &fine_graph, const Partition &coarse_partition,
                                                          const std::vector<int> &vertex_mapping)
{
    std::cout << "    Projecting partition from " << vertex_mapping.size()
              << " coarse vertices to " << fine_graph.n_vertices << " fine vertices" << std::endl;

    // Safety check
    if (vertex_mapping.size() != fine_graph.n_vertices)
    {
        std::cout << "    ERROR: Vertex mapping size mismatch!" << std::endl;
        // Return a simple bisection as fallback
        Partition fallback;
        for (int i = 0; i < fine_graph.n_vertices; i++)
        {
            if (i < fine_graph.n_vertices / 2)
            {
                fallback.left_vertices.push_back(i);
            }
            else
            {
                fallback.right_vertices.push_back(i);
            }
        }
        return fallback;
    }

    // Project partition to finer graph
    Partition projected;

    // Create a mapping from coarse vertex to partition assignment
    std::vector<int> coarse_assignment;
    int max_coarse_vertex = 0;

    // Find the maximum coarse vertex ID
    for (int mapping : vertex_mapping)
    {
        max_coarse_vertex = std::max(max_coarse_vertex, mapping);
    }

    coarse_assignment.resize(max_coarse_vertex + 1, -1);

    // Assign partitions to coarse vertices
    for (int v : coarse_partition.left_vertices)
    {
        if (v >= 0 && v < coarse_assignment.size())
        {
            coarse_assignment[v] = 0; // left partition
        }
    }
    for (int v : coarse_partition.right_vertices)
    {
        if (v >= 0 && v < coarse_assignment.size())
        {
            coarse_assignment[v] = 1; // right partition
        }
    }
    for (int v : coarse_partition.separator_vertices)
    {
        if (v >= 0 && v < coarse_assignment.size())
        {
            coarse_assignment[v] = 2; // separator
        }
    }

    // Project each fine vertex based on its coarse vertex assignment
    for (int fine_v = 0; fine_v < fine_graph.n_vertices; fine_v++)
    {
        int coarse_v = vertex_mapping[fine_v];

        if (coarse_v >= 0 && coarse_v < coarse_assignment.size())
        {
            int assignment = coarse_assignment[coarse_v];
            if (assignment == 0)
            {
                projected.left_vertices.push_back(fine_v);
            }
            else if (assignment == 1)
            {
                projected.right_vertices.push_back(fine_v);
            }
            else if (assignment == 2)
            {
                projected.separator_vertices.push_back(fine_v);
            }
            else
            {
                // Unassigned coarse vertex, default to left
                projected.left_vertices.push_back(fine_v);
            }
        }
        else
        {
            // Invalid coarse vertex mapping, default to left
            projected.left_vertices.push_back(fine_v);
        }
    }

    std::cout << "    Projected partition: L=" << projected.left_vertices.size()
              << " R=" << projected.right_vertices.size()
              << " S=" << projected.separator_vertices.size() << std::endl;

    // Apply basic local refinement (simplified and safer)
    return apply_local_refinement(fine_graph, projected);
}

// Apply local refinement to improve partition quality
Partition ImprovedGPUNestedDissection::apply_local_refinement(const Graph &graph, const Partition &initial_partition)
{
    Partition refined = initial_partition;

    // Copy graph to host for refinement
    std::vector<int> h_row_ptr(graph.n_vertices + 1);
    std::vector<int> h_col_idx(graph.n_edges);
    std::vector<int> h_edge_weights(graph.n_edges);

    hipMemcpy(h_row_ptr.data(), graph.row_ptr,
               (graph.n_vertices + 1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_col_idx.data(), graph.col_idx,
               graph.n_edges * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_edge_weights.data(), graph.edge_weights,
               graph.n_edges * sizeof(int), hipMemcpyDeviceToHost);

    // Create partition assignment array
    std::vector<int> partition(graph.n_vertices, -1);
    for (int v : refined.left_vertices)
        partition[v] = 0;
    for (int v : refined.right_vertices)
        partition[v] = 1;
    for (int v : refined.separator_vertices)
        partition[v] = -1;

    // Simple boundary refinement - move boundary vertices to minimize cut
    bool improved = true;
    int iterations = 0;
    const int max_iterations = 5;

    while (improved && iterations < max_iterations)
    {
        improved = false;
        iterations++;

        // Check each boundary vertex
        for (int v = 0; v < graph.n_vertices; v++)
        {
            if (partition[v] == -1)
                continue; // Skip separator vertices

            // Check if this vertex is on the partition boundary
            bool is_boundary = false;
            for (int e = h_row_ptr[v]; e < h_row_ptr[v + 1]; e++)
            {
                int neighbor = h_col_idx[e];
                if (partition[neighbor] != partition[v] && partition[neighbor] != -1)
                {
                    is_boundary = true;
                    break;
                }
            }

            if (!is_boundary)
                continue;

            // Compute gain of moving this vertex to the other partition
            int current_partition = partition[v];
            int other_partition = 1 - current_partition;

            int internal_weight = 0, external_weight = 0;

            for (int e = h_row_ptr[v]; e < h_row_ptr[v + 1]; e++)
            {
                int neighbor = h_col_idx[e];
                int weight = h_edge_weights[e];

                if (partition[neighbor] == current_partition)
                {
                    internal_weight += weight;
                }
                else if (partition[neighbor] == other_partition)
                {
                    external_weight += weight;
                }
            }

            int gain = external_weight - internal_weight;

            // Move vertex if gain is positive (reduces cut)
            if (gain > 0)
            {
                partition[v] = other_partition;
                improved = true;
            }
        }
    }

    // Rebuild partition from refined assignment
    refined.left_vertices.clear();
    refined.right_vertices.clear();
    // Keep existing separator vertices

    for (int v = 0; v < graph.n_vertices; v++)
    {
        if (partition[v] == 0)
        {
            refined.left_vertices.push_back(v);
        }
        else if (partition[v] == 1)
        {
            refined.right_vertices.push_back(v);
        }
    }

    return refined;
}

// Compute partition cut weight
int ImprovedGPUNestedDissection::compute_partition_cut(const Graph &graph, const std::vector<int> &partition)
{
    // Copy graph data to host
    std::vector<int> h_row_ptr(graph.n_vertices + 1);
    std::vector<int> h_col_idx(graph.n_edges);
    std::vector<int> h_edge_weights(graph.n_edges);

    hipMemcpy(h_row_ptr.data(), graph.row_ptr,
               (graph.n_vertices + 1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_col_idx.data(), graph.col_idx,
               graph.n_edges * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_edge_weights.data(), graph.edge_weights,
               graph.n_edges * sizeof(int), hipMemcpyDeviceToHost);

    int cut = 0;
    for (int v = 0; v < graph.n_vertices; v++)
    {
        for (int e = h_row_ptr[v]; e < h_row_ptr[v + 1]; e++)
        {
            int neighbor = h_col_idx[e];
            if (v < neighbor && partition[v] != partition[neighbor] &&
                partition[v] != -1 && partition[neighbor] != -1)
            {
                cut += h_edge_weights[e];
            }
        }
    }
    return cut;
}

// Identify separator vertices based on partition
void ImprovedGPUNestedDissection::identify_separator_vertices(const Graph &graph, const std::vector<int> &partition,
                                                              std::vector<bool> &is_separator)
{
    // Copy graph data to host
    std::vector<int> h_row_ptr(graph.n_vertices + 1);
    std::vector<int> h_col_idx(graph.n_edges);

    hipMemcpy(h_row_ptr.data(), graph.row_ptr,
               (graph.n_vertices + 1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_col_idx.data(), graph.col_idx,
               graph.n_edges * sizeof(int), hipMemcpyDeviceToHost);

    for (int v = 0; v < graph.n_vertices; v++)
    {
        bool has_left_neighbor = false, has_right_neighbor = false;

        for (int e = h_row_ptr[v]; e < h_row_ptr[v + 1]; e++)
        {
            int neighbor = h_col_idx[e];
            if (partition[neighbor] == 0)
                has_left_neighbor = true;
            if (partition[neighbor] == 1)
                has_right_neighbor = true;
        }

        is_separator[v] = has_left_neighbor && has_right_neighbor;
    }
}

// Static method to create test grid graph
Graph ImprovedGPUNestedDissection::create_test_grid_graph(int grid_size)
{
    const int n_vertices = grid_size * grid_size;

    std::vector<std::vector<std::pair<int, int>>> adj_list(n_vertices);

    // Create 2D grid with edge weights
    for (int i = 0; i < grid_size; i++)
    {
        for (int j = 0; j < grid_size; j++)
        {
            int v = i * grid_size + j;

            // Right neighbor
            if (j < grid_size - 1)
            {
                int neighbor = i * grid_size + (j + 1);
                int weight = 1 + (i + j) % 3; // Varying edge weights
                adj_list[v].push_back({neighbor, weight});
                adj_list[neighbor].push_back({v, weight});
            }

            // Bottom neighbor
            if (i < grid_size - 1)
            {
                int neighbor = (i + 1) * grid_size + j;
                int weight = 1 + (i * j) % 3; // Varying edge weights
                adj_list[v].push_back({neighbor, weight});
                adj_list[neighbor].push_back({v, weight});
            }
        }
    }

    // Count total edges
    int total_edges = 0;
    for (const auto &neighbors : adj_list)
    {
        total_edges += neighbors.size();
    }

    // Create CSR format
    Graph graph(n_vertices, total_edges);
    std::vector<int> h_row_ptr(n_vertices + 1, 0);
    std::vector<int> h_col_idx(total_edges);
    std::vector<int> h_edge_weights(total_edges);
    std::vector<int> h_vertex_weights(n_vertices, 1);

    int edge_pos = 0;
    for (int v = 0; v < n_vertices; v++)
    {
        h_row_ptr[v] = edge_pos;
        for (auto [neighbor, weight] : adj_list[v])
        {
            h_col_idx[edge_pos] = neighbor;
            h_edge_weights[edge_pos] = weight;
            edge_pos++;
        }
    }
    h_row_ptr[n_vertices] = edge_pos;

    // Copy to device
    hipMemcpy(graph.row_ptr, h_row_ptr.data(), (n_vertices + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(graph.col_idx, h_col_idx.data(), total_edges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(graph.edge_weights, h_edge_weights.data(), total_edges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(graph.vertex_weights, h_vertex_weights.data(), n_vertices * sizeof(int), hipMemcpyHostToDevice);

    return graph;
}